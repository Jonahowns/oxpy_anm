#include "hip/hip_runtime.h"
/*
 * CUDASimpleVerletList.cu
 *
 *  Created on: 29/set/2010
 *      Author: lorenzo
 */

#include "CUDASimpleVerletList.h"
#include "CUDA_simple_verlet.cuh"
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include "../../Utilities/oxDNAException.h"

CUDASimpleVerletList::CUDASimpleVerletList() :
				_max_density_multiplier(1) {
	_cells_kernel_cfg.threads_per_block = 0;
	_use_edge = false;
	_N_cells = _old_N_cells = N_edges = -1;
}

CUDASimpleVerletList::~CUDASimpleVerletList() {

}

void CUDASimpleVerletList::clean() {
	if(_d_cells != nullptr) {
		CUDA_SAFE_CALL(hipFree(_d_cells));
		CUDA_SAFE_CALL(hipFree(_d_counters_cells));
		CUDA_SAFE_CALL(hipFree(d_matrix_neighs));
		CUDA_SAFE_CALL(hipFree(d_number_neighs));
		CUDA_SAFE_CALL(hipHostFree(_d_cell_overflow));
	}

	if(_use_edge && d_edge_list != nullptr) {
		CUDA_SAFE_CALL(hipFree(d_edge_list));
		CUDA_SAFE_CALL(hipFree(_d_number_neighs_no_doubles));
	}
}

void CUDASimpleVerletList::get_settings(input_file &inp) {
	getInputBool(&inp, "cells_auto_optimisation", &_auto_optimisation, 0);
	getInputNumber(&inp, "verlet_skin", &_verlet_skin, 1);
	getInputNumber(&inp, "max_density_multiplier", &_max_density_multiplier, 0);
	getInputBool(&inp, "use_edge", &_use_edge, 0);
	if(_use_edge) {
		OX_LOG(Logger::LOG_INFO, "Using edge-based approach");
	}
}

void CUDASimpleVerletList::_init_cells() {
	c_number4 box_sides_n4 = _h_cuda_box->box_sides();
	c_number box_sides[3] = { box_sides_n4.x, box_sides_n4.y, box_sides_n4.z };
	c_number max_factor = pow(2. * _N / _h_cuda_box->V(), 1. / 3.);

	for(int i = 0; i < 3; i++) {
		_N_cells_side[i] = (int) (floor(box_sides[i] / sqrt(_sqr_rverlet)) + 0.1);
		if(_N_cells_side[i] < 3) {
			_N_cells_side[i] = 3;
		}
		if(_auto_optimisation && _N_cells_side[i] > ceil(max_factor * box_sides[i])) {
			_N_cells_side[i] = ceil(max_factor * box_sides[i]);
		}
	}

	_N_cells = _N_cells_side[0] * _N_cells_side[1] * _N_cells_side[2];

	c_number V_cell = _h_cuda_box->V() / (c_number) _N_cells;
	c_number density = _N / _h_cuda_box->V();
	if(density < 0.1) {
		density = 0.1;
	}
	c_number density_factor = density * 5. * _max_density_multiplier;
	_max_N_per_cell = (int) (V_cell * density_factor);
	if(_max_N_per_cell > _N) {
		_max_N_per_cell = _N;
	}
	if(_max_N_per_cell < 5) {
		_max_N_per_cell = 5;
	}

	if(_old_N_cells != -1 && _N_cells != _old_N_cells) {
		CUDA_SAFE_CALL(hipFree(_d_cells));
		CUDA_SAFE_CALL(hipFree(_d_counters_cells));
		_d_cells = _d_counters_cells = nullptr;
		OX_DEBUG("Re-allocating cells on GPU, from %d to %d\n", _old_N_cells, _N_cells);
	}

	if(_d_cells == nullptr) {
		CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_counters_cells, (size_t ) _N_cells * sizeof(int)));
		CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_cells, (size_t ) _N_cells * _max_N_per_cell * sizeof(int)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(verlet_N_cells_side), _N_cells_side, 3 * sizeof(int)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(verlet_max_N_per_cell), &_max_N_per_cell, sizeof(int)));
	}

	_old_N_cells = _N_cells;
}

void CUDASimpleVerletList::init(int N, c_number rcut, CUDABox*h_cuda_box, CUDABox*d_cuda_box) {
	CUDABaseList::init(N, rcut, h_cuda_box, d_cuda_box);

	c_number rverlet = rcut + 2 * _verlet_skin;
	_sqr_rverlet = SQR(rverlet);
	_sqr_verlet_skin = SQR(_verlet_skin);
	_vec_size = N * sizeof(c_number4);

	// volume of a sphere whose radius is ceil(rverlet) times the maximum density (sqrt(2)).
	c_number density = N / h_cuda_box->V();
	if(density < 0.1) density = 0.1;
	c_number density_factor = density * 5. * _max_density_multiplier;
	_max_neigh = (int) ((4 * M_PI * pow(ceil(rverlet), 3) / 3.) * density_factor);
	if(_max_neigh >= N) _max_neigh = N - 1;

	_init_cells();

	OX_LOG(Logger::LOG_INFO, "CUDA max_neigh: %d, max_N_per_cell: %d, N_cells: %d (per side: %d %d %d)", _max_neigh, _max_N_per_cell, _N_cells, _N_cells_side[0], _N_cells_side[1], _N_cells_side[2]);
	OX_LOG(Logger::LOG_INFO, "CUDA Cells mem: %.2lf MBs, lists mem: %.2lf MBs", (double) _N_cells*(1 + _max_N_per_cell) * sizeof(int)/1048576., (double) _N * (1 + _max_neigh) * sizeof(int)/1048576.);

	CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&d_number_neighs, (size_t ) _N * sizeof(int)));
	CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&d_matrix_neighs, (size_t ) _N * _max_neigh * sizeof(int)));

	CUDA_SAFE_CALL(hipHostMalloc(&_d_cell_overflow, sizeof(bool), hipHostMallocDefault));
	_d_cell_overflow[0] = false;

	if(_use_edge) {
		CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&d_edge_list, (size_t ) _N * _max_neigh * sizeof(edge_bond)));
		CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_number_neighs_no_doubles, (size_t ) (_N + 1) * sizeof(int)));
	}

	if(_cells_kernel_cfg.threads_per_block == 0) _cells_kernel_cfg.threads_per_block = 64;
	_cells_kernel_cfg.blocks.x = _N / _cells_kernel_cfg.threads_per_block + ((_N % _cells_kernel_cfg.threads_per_block == 0) ? 0 : 1);
	_cells_kernel_cfg.blocks.y = _cells_kernel_cfg.blocks.z = 1;

	OX_DEBUG("Cells kernel cfg: threads_per_block = %d, blocks = (%d, %d, %d)", _cells_kernel_cfg.threads_per_block,
	_cells_kernel_cfg.blocks.x, _cells_kernel_cfg.blocks.y, _cells_kernel_cfg.blocks.z);

	float f_copy = _sqr_rverlet;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(verlet_sqr_rverlet), &f_copy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(verlet_N), &_N, sizeof(int)));
}

void CUDASimpleVerletList::update(c_number4 *poss, c_number4 *list_poss, LR_bonds *bonds) {
	_init_cells();
	CUDA_SAFE_CALL(hipMemset(_d_counters_cells, 0, _N_cells * sizeof(int)));

	// fill cells
	simple_fill_cells
		<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
		(poss, _d_cells, _d_counters_cells, _d_cell_overflow, _d_cuda_box);
	CUT_CHECK_ERROR("fill_cells (SimpleVerlet) error");

	hipDeviceSynchronize();
	if(_d_cell_overflow[0] == true) {
		throw oxDNAException("A cell contains more than _max_n_per_cell (%d) particles. Please increase the value of max_density_multiplier (which defaults to 1) in the input file\n", _max_N_per_cell);
	}

	// texture binding for the number of particles contained in each cell
	hipBindTexture(0, counters_cells_tex, _d_counters_cells, sizeof(int) * _N_cells);

	// for edge based approach
	if(_use_edge) {
		edge_update_neigh_list
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(poss, list_poss, _d_cells, d_matrix_neighs, d_number_neighs, _d_number_neighs_no_doubles, bonds, _d_cuda_box);
		CUT_CHECK_ERROR("edge_update_neigh_list (SimpleVerlet) error");

		// thrust operates on the GPU
		thrust::device_ptr<int> d_number_neighs_no_doubles_w(_d_number_neighs_no_doubles);
		d_number_neighs_no_doubles_w[_N] = 0;
		thrust::exclusive_scan(d_number_neighs_no_doubles_w, d_number_neighs_no_doubles_w + _N + 1, d_number_neighs_no_doubles_w);
		N_edges = d_number_neighs_no_doubles_w[_N];
		// get edge list from matrix_neighs
		compress_matrix_neighs
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(d_matrix_neighs, d_number_neighs, _d_number_neighs_no_doubles, d_edge_list);
		CUT_CHECK_ERROR("compress_matrix_neighs error");
	}
	else {
		simple_update_neigh_list
			<<<_cells_kernel_cfg.blocks, _cells_kernel_cfg.threads_per_block>>>
			(poss, list_poss, _d_cells, d_matrix_neighs, d_number_neighs, bonds, _d_cuda_box);
		CUT_CHECK_ERROR("update_neigh_list (SimpleVerlet) error");
	}

	hipUnbindTexture(counters_cells_tex);
}
